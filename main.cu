#include "defs.h"
#include "kernel.cu"
#include <iostream>
#include <cstdio>

using namespace std;

int main()
{
    FILE *fp = fopen("sample.txt","r");
    if(fp == NULL)
        cout<<"File does not exist\n";

    unsigned int *transactions = NULL;
    char *line = NULL;
    size_t len = 0;
    char *ln;
    unsigned int lines = 0;
    transactions = (unsigned int *) malloc(max_num_of_transaction * max_items_in_transaction * sizeof(unsigned int));

    int offset = 0;
    while (getline(&line, &len, fp) != -1 && lines < max_num_of_transaction){
        ln = strtok(line, " ");
        int i = offset;
        while(ln != NULL)
        {
            cout<<(int)ln[0]<<" ";
            transactions[i++] = (unsigned int)ln[0];
            ln = strtok(NULL, " ");
        }
        offset += max_items_in_transaction;
        cout<<endl;
    }

    unsigned int *d_transactions;
    //hipError_t cuda_ret;
    
    hipMalloc((void**)&d_transactions, max_num_of_transaction * max_items_in_transaction * sizeof(unsigned int));
    hipMemcpy(d_transactions, transactions , max_num_of_transaction * max_items_in_transaction * sizeof(unsigned int), hipMemcpyHostToDevice);

    struct node *d_output,*h_output;
    h_output = (node *) malloc(max_items_in_transaction * sizeof(node));
    hipMalloc((void**)&d_output, max_items_in_transaction * sizeof(node));
    hipMemset(d_output, 0 , max_items_in_transaction * sizeof(node));

    dim3 grid_dim = ((max_num_of_transaction * max_items_in_transaction) - 1)/max_items_in_transaction + 1;
    dim3 block_dim = max_items_in_transaction;

    generate_fp_tree <<<grid_dim,block_dim>>> (d_transactions,d_output);

    hipMemcpy(h_output, d_output , max_items_in_transaction * sizeof(node), hipMemcpyDeviceToHost);

    cout<<"Original output:\n";
    for(int i=0;i<5;i++)
    {
        for(int j=0;j<5;j++)
            cout<<h_output[i].items[j]<<" ";
        cout<<endl;
    }

}

