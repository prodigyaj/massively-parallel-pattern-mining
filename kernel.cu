
#include <hip/hip_runtime.h>
struct node
{
    int items[10];
    int parent[10];
};

__global__ void generate_fp_tree(unsigned int* input, node *output)
{
    int tx = threadIdx.x;
    int gtx = blockIdx.x * blockDim.x + threadIdx.x;
    if(input[gtx]!=0)
        atomicAdd(&output[tx].items[input[gtx]-65],1);
        //output[tx].items[input[gtx]-65]+=1;
    //output[tx].items[0]+=input[gtx];
}
